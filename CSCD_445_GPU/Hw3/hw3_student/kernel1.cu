#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];
    // TODO, implement this kernel below

    // global thread(data) row index 
  unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
  i = i + 1; //because the edge of the data is not processed
  
  // global thread(data) column index
  unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
  j = j + 1; //because the edge of the data is not processed  

  // assign value of thread into corresponding index of s_data to be referred to for math below

  s_data[i * pitch + j] = g_dataA[i * pitch + j];

  __syncthreads();

  // check the boundary
  if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) return;

  g_dataB[i * pitch + j] = (
                              0.2f * s_data[i * pitch + j] +               //itself
                              0.1f * s_data[(i-1) * pitch +  j   ] +       //N
                              0.1f * s_data[(i-1) * pitch + (j+1)] +       //NE
                              0.1f * s_data[ i    * pitch + (j+1)] +       //E
                              0.1f * s_data[(i+1) * pitch + (j+1)] +       //SE
                              0.1f * s_data[(i+1) * pitch +  j   ] +       //S
                              0.1f * s_data[(i+1) * pitch + (j-1)] +       //SW
                              0.1f * s_data[ i    * pitch + (j-1)] +       //W
                              0.1f * s_data[(i-1) * pitch + (j-1)]         //NW
                           ) * 0.95f;
}


