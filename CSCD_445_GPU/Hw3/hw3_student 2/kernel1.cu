#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];
    // TODO, implement this kernel below
    
     unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    i = i + 1; //because the edge of the data is not processed
  
    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    j = j + 1; //because the edge of the data is not processed  


    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) return;

    int IDX = threadIdx.x;
    int IDY = 1;
    
    ////////////////////////////////////////////////////////////////////////////////
    // For thread 0
    ////////////////////////////////////////////////////////////////////////////////
    if(IDX == 0){
        s_data[IDX] = g_dataA[(i-1) * floatpitch + (j-1)];//Top left
        s_data[(blockDim.x+2)+IDX+1] = g_dataA[ i    * floatpitch + (j-1)];//Left
        s_data[(IDY+1)*(blockDim.x+2)+IDX+1] = g_dataA[(i+1) * floatpitch + (j-1)]; //Bottom left
        s_data[IDX+1] = g_dataA[(i-1) * floatpitch +  j   ]; //Top
        s_data[(blockDim.x+2)+IDX+2] = g_dataA[i * floatpitch + j]; //Itself
        s_data[(IDY+1)*(blockDim.x+2)+IDX+2] = g_dataA[(i+1) * floatpitch +  j   ]; //Bottom   
    }else if(IDX == ((width-2)%blockDim.x)-1 && j == width-2 || IDX == blockDim.x-1){ //<-Boundary check
    ////////////////////////////////////////////////////////////////////////////////
    // For farthest left thread within width
    ////////////////////////////////////////////////////////////////////////////////
        s_data[IDX+2] = g_dataA[(i-1) * floatpitch + (j+1)];//Top right
        s_data[(blockDim.x+2)+IDX+3]  = g_dataA[ i    * floatpitch + (j+1)]; //Right
        s_data[((IDY+1)*(blockDim.x+2))+IDX+3] = g_dataA[(i+1) * floatpitch + (j+1)] ; //Bottom right
        s_data[IDX+1] = g_dataA[(i-1) * floatpitch +  j   ]; //Top
        s_data[(blockDim.x+2)+IDX+2] = g_dataA[i * floatpitch + j]; //Itself
        s_data[(IDY+1)*(blockDim.x+2)+IDX+2] = g_dataA[(i+1) * floatpitch +  j   ]; //Bottom
    }else{
    ////////////////////////////////////////////////////////////////////////////////
    // For all middle threads
    ////////////////////////////////////////////////////////////////////////////////
        s_data[IDX+1] = g_dataA[(i-1) * floatpitch +  j   ]; //Top
        s_data[(blockDim.x+2)+IDX+2] = g_dataA[i * floatpitch + j]; //Itself
        s_data[(IDY+1)*(blockDim.x+2)+IDX+2] = g_dataA[(i+1) * floatpitch +  j   ]; //Bottom
    }
    __syncthreads(); //Make sure all threads move the mem into shared before moving on
    
   
    g_dataB[i * floatpitch + j] = (
                    0.2f * s_data[(blockDim.x+2)+IDX+2] +               //itself
                    0.1f * s_data[IDX+1] +       //N ABOVE
                    0.1f * s_data[IDX+2] +       //NE ABOVE RIGHT
                    0.1f * s_data[(blockDim.x+2)+IDX+3] +       //E RIGHT
                    0.1f * s_data[((IDY+1)*(blockDim.x+2))+IDX+3] +       //SE BOTTOM RIGHT
                    0.1f * s_data[(IDY+1)*(blockDim.x+2)+IDX+2] +       //S BOTTOM
                    0.1f * s_data[(IDY+1)*(blockDim.x+2)+IDX+1] +       //SW BOTTOM LEFT 
                    0.1f * s_data[(blockDim.x+2)+IDX+1] +       //W LEFT 
                    0.1f * s_data[IDX]         //NW TOP LEFT 
                ) * 0.95f;

 



}

