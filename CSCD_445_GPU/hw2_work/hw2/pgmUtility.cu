#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

__host__ int *pgmRead(char **header, int *numRows, int *numCols, FILE *in){
    int i, j;
    
    // read in header of the image first
    for( i = 0; i < rowsInHeader; i ++)
    {
        if ( header[i] == NULL )
        {
            return NULL;
        }
        if( fgets( header[i], maxSizeHeadRow, in ) == NULL )
        {
            return NULL;
        }
    }

    // extract rows of pixels and columns of pixels
    puts(header[rowsInHeader - 2]);
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows );  // in pgm the first number is # of cols
  
    int *pixels = (int*)malloc((*numCols) * (*numRows) * sizeof(int));

    
    // read in all pixels into the pixels array.

    for( i = 0; i < *numRows; i ++ )
        for( j = 0; j < *numCols; j ++ )
            if(fscanf(in, "%d ", &pixels[(i*(*numCols))+j]) < 0 )
                return NULL;

    
    return pixels;
}

__host__ int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ){
    
    int num_bytes = (numCols*numRows*sizeof(int));
    int *d_a;
    hipMalloc((void**)&d_a, num_bytes);
    hipMemcpy(d_a, pixels, num_bytes, hipMemcpyHostToDevice);

    dim3 grid, block;
    block.x = 16;
    block.y = 16;
    grid.x = ceil((float)numCols/block.x);
    grid.y = ceil((float)numRows/block.y);

    pgmDrawCircleK<<<grid, block>>>(d_a, numRows, numCols, centerRow, centerCol, radius);

    hipMemcpy(pixels, d_a, num_bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);

    return 0;
}

__host__ int pgmDrawEdge( int *pixels, int numRows, int numCols, int edgeWidth, char **header ){
    int num_bytes = (numCols*numRows*sizeof(int));
    int *d_a;

    hipMalloc((void**)&d_a, num_bytes);
    hipMemcpy(d_a, pixels, num_bytes, hipMemcpyHostToDevice);

    dim3 grid, block;
    block.x = 16;
    block.y = 16;
    grid.x = ceil((float)numCols/block.x);
    grid.y = ceil((float)numRows/block.y);

    pgmDrawEdgeK<<<grid,block>>>(d_a, numRows, numCols, edgeWidth);

    hipMemcpy(pixels, d_a, num_bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);

    return 0;
}

__host__ int pgmDrawLine( int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ){
    int num_bytes = (numCols*numRows*sizeof(int));
    int *d_a;

    hipMalloc((void**)&d_a, num_bytes);
    hipMemcpy(d_a, pixels, num_bytes, hipMemcpyHostToDevice);

    dim3 grid, block;
    block.x = 16;
    block.y = 16;
    grid.x = ceil((float)numCols/block.x);
    grid.y = ceil((float)numRows/block.y);

    pgmDrawLineK<<<grid, block>>>(d_a, numRows, numCols, p1row, p1col, p2row, p2col);

    hipMemcpy(pixels, d_a, num_bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);

    return 0;
}

__host__ int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out )
{
    int i, j;
    // write the header
    for ( i = 0; i < rowsInHeader; i ++ )
    {
        fprintf(out, "%s", *( header + i ) );
    }
    
    // write the pixels
    for( i = 0; i < numRows; i ++ )
    {
        for ( j = 0; j < numCols; j ++ )
        {
            if ( j < numCols - 1 )
                fprintf(out, "%d ", pixels[(i*numCols)+j]);
            else
                fprintf(out, "%d\n", pixels[(i*numCols)+j]);
        }
    }
    return 0;
}

