

/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */

#include <hip/hip_runtime.h>
#include <math.h>
__device__ float distance( int p1x, int p1y, int p2x, int p2y)
{
    return sqrt( (float)(((p1x - p2x) * (p1x - p2x)) + ((p1y - p2y)*(p1y - p2y))) );
}

__global__ void pgmDrawCircleK(int *d_pixels, int numRows, int numCols, int centerRow, int centerCol, int redius){
    
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int r = redius*redius;

    if((Row < numRows) && (Col < numCols)){
        if( ((Col-centerCol)*(Col-centerCol))+((Row-centerRow)*(Row-centerRow)) < r){
            d_pixels[Row*numCols+Col] = 0;
        }
    }
}

__global__ void pgmDrawEdgeK( int *pixels, int numRows, int numCols, int edgeWidth)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    

    if((Row < numRows) && (Col < numCols)){
        if((Row < edgeWidth | Col <edgeWidth) | (edgeWidth-Row)+numRows < edgeWidth*2 | (edgeWidth-Col)+numCols < edgeWidth*2 )
                pixels[(Row*(numCols))+Col] = 0;
    }
}

__global__ void pgmDrawLineK( int *pixels, int numRows, int numCols, int p1row, int p1col, int p2row, int p2col ){
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    double slope = (double)(p2row-p1row)/(p2col-p1col);
    //double b = (double)p2row-((slope)*p2col);

    double c = p1row - slope * p1col;
    double yInt = slope * 0 +c;

     if((Row >= p1row) && (Row < p2row) && (Col < numCols)){
         if( fabs((slope * Col + yInt)-(double)Row) < .5){
                pixels[(Row*(numCols))+Col] = 0;
            } 
     }
}